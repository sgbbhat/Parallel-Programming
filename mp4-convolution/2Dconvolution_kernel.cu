#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#define TILE_SIZE 12

#include <stdio.h>
#include "2Dconvolution.h"
#include <sys/time.h>

// Matrix convolution kernel specification
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{
	__shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
        
	int tx = threadIdx.x;
        int ty = threadIdx.y;
        
	int col_o = blockIdx.x * TILE_SIZE + tx;
        int row_o = blockIdx.y * TILE_SIZE + ty;
        
	int row_i = row_o - 2;
        int col_i = col_o - 2;
        
	float output = 0.0f;
	
	// Loading input
        if((row_i >= 0) && (row_i < N.height) && (col_i >= 0)  && (col_i < N.width))
        {
                Nds[ty][tx] = N.elements[row_i*N.width + col_i];
        }
        else
        {
                Nds[ty][tx] = 0.0f;
        }
        __syncthreads();

	// Handling boundary condition
        if(ty < TILE_SIZE && tx < TILE_SIZE)
        {
                for(int i = 0; i < KERNEL_SIZE; i++)
                {
                        for(int j = 0; j < KERNEL_SIZE; j++)
                        {
                                output += Mc[i][j] *  Nds[i+ty][j+tx] ;
                        }
                }
	if(row_o < P.height && col_o < P.width)
                P.elements[row_o * P.width + col_o] = output;
        }
	__syncthreads();
}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
